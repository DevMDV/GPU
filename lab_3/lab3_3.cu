
#include <hip/hip_runtime.h>
#include "cstdio"
#include <iostream>
#include <chrono>

constexpr size_t SIZE = 16384 * 3; // 16384 * 3
constexpr size_t BLOCK_COUNT = 4096; //for shared alg 16384 * 3 for simple
constexpr size_t BLOCK_SIZE = SIZE / BLOCK_COUNT;
constexpr size_t THREAD_PER_BLOCK = 128;


/*
 * GPU Elapsed time 23.9948
 * CPU Elapsed time 6720
 */

template<typename T>
__global__ void sumMatrixRowShared(T* matrix, T* result)
{
  __shared__ float data[THREAD_PER_BLOCK];
  for(size_t row_num = BLOCK_SIZE * blockIdx.x;
      row_num < BLOCK_SIZE * (blockIdx.x + 1); row_num++) {
    size_t row_start = row_num * SIZE;
    size_t idx = threadIdx.x;
    data[idx] = matrix[row_start + idx];
    for (size_t i=1; i * THREAD_PER_BLOCK + idx < SIZE; i++)
    {
      data[idx] = data[idx] + matrix[row_start + i * THREAD_PER_BLOCK + idx];
    }
    __syncthreads();
    for (size_t s = 1; s < blockDim.x; s <<= 1 )
    {
      size_t index = 2 * s * idx;
      if ( index < blockDim.x )
        data [index] += data [index + s];
      __syncthreads ();
    }

    if (idx == 0)
      result[row_num] = data[0];
  }
}

template <typename T>
void sumMatrixRowCPU(const float* matrix, T* result)
{
  for(int idx = 0; idx < SIZE; idx++)
  {
    result[idx] = 0;
    for(size_t i=0; i < SIZE; i++)
    {
      result[idx] = result[idx] + matrix[idx * SIZE + i];
    }
  }
}

__host__ int main()
{
  //Выделяем память под вектора
  auto* matrix = new float[SIZE * SIZE];
  auto* result = new float[SIZE];
  auto* result_1 = new float[SIZE];
  //Инициализируем значения векторов
  for (int i = 0; i < SIZE * SIZE; i++)
  {
    matrix[i] = int(i/SIZE);
    result[i%SIZE] = 0;
    result_1[i%SIZE] = 0;
  }
  float* gpu_matrix;
  float* gpu_result;
  //Выделяем память для векторов на видеокарте
  hipMalloc((void**)&gpu_matrix, sizeof(float) * SIZE * SIZE);
  hipMalloc((void**)&gpu_result, sizeof(float) * SIZE);
  hipMemcpy(gpu_matrix, matrix, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice);
  hipMemcpy(gpu_result, result, sizeof(float) * SIZE, hipMemcpyHostToDevice);

  dim3 gridSize = dim3(BLOCK_COUNT, 1, 1); //Размер используемой сетки
  dim3 blockSize = dim3(THREAD_PER_BLOCK, 1, 1); //Размер используемого блока
  //Выполняем вызов функции ядра

  hipEvent_t kernel_start;
  hipEventCreate(&kernel_start);
  hipEventRecord(kernel_start, 0);

  sumMatrixRowShared<<<gridSize, blockSize>>>(gpu_matrix, gpu_result);

  hipEvent_t syncEvent; //Дескриптор события
  hipEventCreate(&syncEvent); //Создаем event
  hipEventRecord(syncEvent, 0); //Записываем event
  hipEventSynchronize(syncEvent); //Синхронизируем event
  float time;
  hipEventElapsedTime(&time, kernel_start, syncEvent);

  hipMemcpy(result, gpu_result, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

  std::cout << "GPU Elapsed time " << time << std::endl;

  auto t1 = std::chrono::high_resolution_clock::now();
  sumMatrixRowCPU(matrix, result_1);
  auto t2 = std::chrono::high_resolution_clock::now();

  std::cout << "CPU Elapsed time " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() <<std::endl;

  for (int i = 0; i < 10; i++)
  {
    printf("Element #%i: %.1f %1.f\n", i , result[i], result_1[i]);
  }

  // Освобождаем ресурсы
  hipEventDestroy(syncEvent);
  hipFree(gpu_matrix);
  hipFree(gpu_result);
  delete[] result;
  delete[] matrix;
}
