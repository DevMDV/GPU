#include "cstdio"
#include <iostream>
#include <chrono>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

constexpr size_t SIZE = 16384 * 3; // 16384 * 3
constexpr size_t BLOCK_COUNT = 1024 + 512; //for shared alg 16384 * 3 for simple
constexpr size_t THREAD_PER_BLOCK = SIZE / BLOCK_COUNT;
constexpr size_t BLOCK_SIZE = SIZE / BLOCK_COUNT;
constexpr size_t SHARED_MEMORY = 16384 * 3; // opt 16384 * 3 for shared alg
constexpr size_t SHARED_MEMORY_COUNT = SHARED_MEMORY / sizeof(float);

/*
 * GPU Elapsed time 409.04
 * CPU Elapsed time 6828
*/


template<typename T>
__global__ void sumMatrixRowShared(const float* matrix, T* result)
{
  __shared__ float ss[SHARED_MEMORY / sizeof(T)];
  unsigned int idx = threadIdx.x;
  unsigned int block_idx = blockIdx.x;
  idx = idx + (SIZE/BLOCK_COUNT) * block_idx;
  result[idx] = 0;

  constexpr int SHARED_SIZE = SHARED_MEMORY / sizeof(T) / THREAD_PER_BLOCK;
  for (size_t batch_num=0; batch_num < SIZE / SHARED_SIZE; batch_num++)
  {
    for (size_t i=0; i < SHARED_SIZE; i++)
    {
      ss[SHARED_SIZE * threadIdx.x + i] = matrix[idx * SIZE + batch_num * SHARED_SIZE + i];
    }

    for(size_t i=0; i < SHARED_SIZE; i++)
    {
      result[idx] += ss[SHARED_SIZE * threadIdx.x + i];
    }
  }
}


template <typename T>
void sumMatrixRowCPU(const float* matrix, T* result)
{
  for(int idx = 0; idx < SIZE; idx++)
  {
    result[idx] = 0;
    for(size_t i=0; i < SIZE; i++)
    {
      result[idx] = result[idx] + matrix[idx * SIZE + i];
    }
  }
}

__host__ int main()
{
  //Выделяем память под вектора
  auto* matrix = new float[SIZE * SIZE];
  auto* result = new float[SIZE];
  //Инициализируем значения векторов
  for (int i = 0; i < SIZE * SIZE; i++)
  {
    matrix[i] = int(i/SIZE);
    result[i%SIZE] = 0;
  }

  float* gpu_matrix;
  float* gpu_result;
  //Выделяем память для векторов на видеокарте
  hipMalloc((void**)&gpu_matrix, sizeof(float) * SIZE * SIZE);
  hipMalloc((void**)&gpu_result, sizeof(float) * SIZE);
  hipMemcpy(gpu_matrix, matrix, sizeof(float) * SIZE * SIZE, hipMemcpyHostToDevice);
  hipMemcpy(gpu_result, result, sizeof(float) * SIZE, hipMemcpyHostToDevice);

  dim3 gridSize = dim3(BLOCK_COUNT, 1, 1); //Размер используемой сетки
  dim3 blockSize = dim3(THREAD_PER_BLOCK, 1, 1); //Размер используемого блока
  //Выполняем вызов функции ядра

  hipEvent_t kernel_start;
  hipEventCreate(&kernel_start);
  hipEventRecord(kernel_start, 0);

  sumMatrixRowShared<<<gridSize, blockSize>>>(gpu_matrix, gpu_result);

  hipEvent_t syncEvent; //Дескриптор события
  hipEventCreate(&syncEvent); //Создаем event
  hipEventRecord(syncEvent, 0); //Записываем event
  hipEventSynchronize(syncEvent); //Синхронизируем event
  float time;
  hipEventElapsedTime(&time, kernel_start, syncEvent);

  hipMemcpy(result, gpu_result, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

  //Результаты расчета
  for (int i = 0; i < 10; i++)
  {
    printf("Element #%i: %.1f\n", i , result[i]);
  }
  std::cout << "GPU Elapsed time " << time << std::endl;

  auto t1 = std::chrono::high_resolution_clock::now();
  sumMatrixRowCPU(matrix, result);
  auto t2 = std::chrono::high_resolution_clock::now();

  std::cout << "CPU Elapsed time " << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() <<std::endl;

  for (int i = 0; i < 10; i++)
  {
    printf("Element #%i: %.1f\n", i , result[i]);
  }

  // Освобождаем ресурсы
  hipEventDestroy(syncEvent);
  hipFree(gpu_matrix);
  hipFree(gpu_result);
  delete[] result;
  delete[] matrix;
}
