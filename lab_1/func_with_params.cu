#include "hip/hip_runtime.h"

#include <cstdio>

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
__device__ int  add(int a, int b)
{
  return a + b;
}

__global__ void kernel(int a, int b, int* c)
{
   int q = add(a,b);
   *c = q;
}

int main()
{ int c;
    int *dev_c;
    HANDLE_ERROR (hipMalloc((void**)&dev_c, sizeof(int)));
    kernel<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR (hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    printf ("2 + 7 = %d\n", c);
    hipFree(dev_c);
    return 0;
}

